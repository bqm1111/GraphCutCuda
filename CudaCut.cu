#include "hip/hip_runtime.h"
#include "cudacut.h"
#include "CudaCut_kernel.cu"

CudaCut::CudaCut(int image_width, int image_height, int overlap_width)
    : height(image_height), width(overlap_width), image_width(image_width){
    graph_size = width*height;
    size_int = sizeof(int)*graph_size;


}
__global__ void warm_up_kernel(int width){
    unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int tid = iy*width + ix;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid;
}
void CudaCut::cudaWarmUp(){
    dim3 block(16, 16, 1);
    dim3 grid((width+block.x-1)/block.x, (height+block.y-1)/block.y, 1);
    warm_up_kernel<<<grid, block>>>(width);
    gpuErrChk(hipDeviceSynchronize());
}

void CudaCut::h_mem_init()
{

    h_left_weight = (int*)malloc(size_int);
    h_right_weight = (int*)malloc(size_int);
    h_down_weight  = (int*)malloc(size_int);
    h_up_weight = (int*)malloc(size_int);
    h_graph_height  =  (int*)malloc(sizeof(int)*graph_size);

    h_excess_flow = (int*)malloc(sizeof(int)*graph_size);
    h_relabel_mask = (int*)malloc(size_int);
    h_height_backup = (int*)malloc(4*size_int);

    h_visited = (int*)malloc(sizeof(int)*graph_size);
    h_frontier = (bool*)malloc(sizeof(bool)*graph_size);
    h_visited_backward = (bool*)malloc(sizeof(bool)*graph_size);
    h_visited_forward = (bool*)malloc(sizeof(bool)*graph_size);

    h_m1 = (unsigned char *)malloc(sizeof(unsigned char)*graph_size);
    h_m2 = (unsigned char*)malloc(sizeof(unsigned char)*graph_size);
    h_process_area = (int*)malloc(size_int);
    h_horizontal = (int*)malloc(size_int + height*sizeof(int));
    h_vertical = (int*)malloc(size_int + width*sizeof(int));

    h_bfs_counter = (int*)malloc(sizeof(int)*graph_size);
    h_active_node = (int*)malloc(sizeof(int)*4000);
    memset(h_active_node, 0, sizeof(int)*4000);
    h_average_active = (int*)malloc(sizeof(int)*4000);
    memset(h_average_active, 0, sizeof(int)*4000);

    data_ = (unsigned char *)malloc(sizeof(unsigned char)*graph_size);
    memset(data_, 255, graph_size);



    // initial h_weight, h_flow from input

}

void CudaCut::d_mem_init()
{
    //    gpuErrChk(hipMalloc((void**)&d_left_weight, size_int));
    //    gpuErrChk(hipMalloc((void**)&d_right_weight, size_int));
    //    gpuErrChk(hipMalloc((void**)&d_down_weight, size_int));
    //    gpuErrChk(hipMalloc((void**)&d_up_weight, size_int));

    //gpuErrChk(hipMalloc((void**)&d_graph_height, sizeof(int)*graph_size));
    //    gpuErrChk(hipMalloc((void**)&d_excess_flow, sizeof(int)*graph_size));
    gpuErrChk(hipMalloc((void**)&d_relabel_mask, size_int));
    gpuErrChk(hipMalloc((void**)&d_height_backup, 4*size_int));

    //    gpuErrChk(hipMalloc((void**)&d_visited, sizeof(int)*graph_size));
    //    gpuErrChk(hipMalloc((void**)&d_frontier, sizeof(bool)*graph_size));

    //    gpuErrChk(hipMalloc((void**)&d_m1, size_int));
    //    gpuErrChk(hipMalloc((void**)&d_m2, size_int));
    //    gpuErrChk(hipMalloc((void**)&d_process_area, size_int));
    //gpuErrChk(hipMalloc((void**)&d_horizontal, size_int + height*sizeof(int)));
    //gpuErrChk(hipMalloc((void**)&d_vertical, size_int + width*sizeof(int)));

    //    gpuErrChk(hipMalloc((void**)&d_push_block_position, sizeof(int)*(5*height)));
    gpuErrChk(hipMalloc((void**)&d_up_right_sum, sizeof(int)*graph_size));
    gpuErrChk(hipMalloc((void**)&d_up_left_sum, sizeof(int)*graph_size));
    gpuErrChk(hipMalloc((void**)&d_down_right_sum, sizeof(int)*graph_size));
    gpuErrChk(hipMalloc((void**)&d_down_left_sum, sizeof(int)*graph_size));
    gpuErrChk(hipMalloc((void**)&d_bfs_counter, sizeof(int)*graph_size));


    hipMallocManaged((void**)&d_left_weight, size_int);
    hipMallocManaged((void**)&d_right_weight, size_int);
    hipMallocManaged((void**)&d_down_weight, size_int);
    hipMallocManaged((void**)&d_up_weight, size_int);
    hipMallocManaged((void**)&d_excess_flow, size_int);
    hipMallocManaged((void**)&d_horizontal, size_int + height*sizeof(int));
    hipMallocManaged((void**)&d_vertical,  size_int + width*sizeof(int));
    hipMallocManaged((void**)&d_graph_height, size_int);
    gpuErrChk(hipMallocManaged((void**)&d_min_block, sizeof(int)*height));
    gpuErrChk(hipMallocManaged((void**)&d_min_col, sizeof(int)*width));
}

__global__ void cudaFindMin(int * src, int *dst, int width, int height)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < height)
    {
        int minimum = 1000000;
        for(int y = 1; y < width-1; y++)
        {
            int index = idx * width + y;
            if(src[index] < minimum)
            {
                minimum = src[index];
            }
        }
        //        if(minimum == 0) minimum = 1;
        printf("%d ", minimum);
        dst[idx] = minimum;
    }
}

__global__ void cudaFindMinCol(int * src, int *dst, int width, int height)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < width)
    {
        int minimum = 1000000;
        for(int y = 1; y < height-1; y++)
        {
            int index = y*width + idx;
            if(src[index] < minimum)
            {
                minimum = src[index];
            }
        }
        //        if(minimum == 0) minimum = 1;
        printf("%d ", minimum);
        dst[idx] = minimum;
    }
}

void CudaCut::findMin(int * src, int * dst, int width, int height)
{
    cudaFindMin<<<height / threadPerBlock_x, threadPerBlock_x>>>(src, dst, width, height);
    gpuErrChk(hipDeviceSynchronize());
}

void CudaCut::findMinCol(int * src, int * dst, int width, int height)
{
    cudaFindMinCol<<<width / threadPerBlock_x, threadPerBlock_x>>>(src, dst, width, height);
    gpuErrChk(hipDeviceSynchronize());
}

__global__ void
setupGraph_kernel(int *d_horizontal, int *d_vertical, int *d_right_weight, int *d_left_weight, int *d_up_weight,
                  int *d_down_weight, int *d_excess_flow, int *d_push_block_position, int *d_graph_height,
                  int *d_relabel_mask, int width, int height, int N){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int node_i = iy*width + ix;
    d_right_weight[node_i] = d_horizontal[iy*(width+1) + ix+1];
    d_left_weight[node_i] = d_horizontal[iy*(width+1) + ix];

    d_down_weight[node_i] = d_vertical[(iy+1)*width + ix];
    d_up_weight[node_i] = d_vertical[node_i];
    d_excess_flow[node_i] = 0;
    d_graph_height[node_i] = width - ix - 1;
    d_relabel_mask[node_i] = 0;

    //blockIdx.x == 0? d_push_block_position[iy*5 + blockIdx.x] = 1 : d_push_block_position[iy*5 + blockIdx.x] = 0;
    //    if(blockIdx.x == 0 && threadIdx.x == 1){
    //        d_push_block_position[iy*5 + blockIdx.x] = 1;
    //    }
    //    if(blockIdx.x != 0 && threadIdx.x == 0){
    //        d_push_block_position[iy*5 + blockIdx.x] = 0;
    //    }

}

__global__ void
adjustGraph_kernel(int *d_excess_flow, int *d_left_weight, int *d_right_weight, int *d_down_weight, int*d_up_weight,
                   int *d_graph_height,int *d_up_right_sum, int *d_up_left_sum, int *d_down_right_sum, int *d_down_left_sum,
                   int width, int height, int N){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int node_i = iy*width + ix;

    if(ix == 1){
        int tmp = d_right_weight[node_i];
        d_excess_flow[node_i] = tmp;
        d_left_weight[node_i] = 1<<21;
    }

    if(ix == 0){
        //d_excess_flow[node_i] = -10000;
        d_right_weight[node_i] = 0;
        d_up_weight[node_i] = 0;
        d_down_weight[node_i] = 0;
        d_graph_height[node_i] = N;
    }
    if(ix == width-1){
        d_left_weight[node_i] = 0;
        d_up_weight[node_i] = 0;
        d_down_weight[node_i] = 0;
        d_right_weight[node_i-1] = 1<<21;
    }


    __syncthreads();
    d_up_right_sum[node_i] = d_right_weight[node_i] + d_up_weight[node_i];
    d_up_left_sum[node_i] = d_left_weight[node_i] + d_up_weight[node_i];

    d_down_right_sum[node_i] = d_right_weight[node_i] + d_down_weight[node_i];
    d_down_left_sum[node_i] = d_left_weight[node_i] + d_down_weight[node_i];
    //    if(ix % 16 == 0 && ix != 0){
    //        int tmp = d_left_weight[node_i];
    //        d_excess_flow[node_i] = tmp;
    //        d_left_weight[node_i] = tmp + tmp;
    //        d_right_weight[node_i - 1] = 0;
    //    }

}

__global__ void
adjustGraph_kernel1(int *d_excess_flow, int *d_left_weight, int *d_right_weight, int *d_down_weight, int*d_up_weight,
                    int *d_graph_height, int *d_up_right_sum, int *d_up_left_sum, int *d_down_right_sum, int *d_down_left_sum,
                    int width, int height, int N, int * d_min_block){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    unsigned int node_i = iy*width + ix;
    //printf("cond in push_block_kernel %d\n", cond);
    //int pos = d_push_block_position[iy*gridDim.x + blockIdx.x];
    //printf("pos of blockIdx.%d  %d\n", blockIdx.x, pos);
    //__shared__ int flow_block[10];
    {
        __shared__ int smem[threadPerBlock_x * threadPerBlock_y];
        int idx = (threadIdx.y * threadPerBlock_x) + threadIdx.x;
        int min;
        //        if(d_right_weight[node_i] >0 && d_down_weight[node_i] > 0)
        //        {
        //            d_left_weight[node_i] = 0;
        //            d_up_weight[node_i] = 0;
        //        }
        //        if(d_right_weight[node_i] == d_min_block[iy])
        //        {
        //            d_left_weight[node_i] += d_min_block[iy];
        //            d_right_weight[node_i] += d_min_block[iy];
        //        }
        //        if(ix > 1 /*&& ix < width-threadPerBlock_x*3*/ /*&& (ix % threadPerBlock_x == 0)*/)
        //        {
        ////            if(d_right_weight[node_i] == d_min_block[iy])
        ////            {
        ////                d_right_weight[node_i] = 0;
        ////                d_down_weight[node_i] = 0;
        ////            }
        //            //            printf("%d - %d\n", node_i, d_min_block[iy]);
        ////                        if(d_min_block[iy] == 0)
        ////                        {
        ////                            d_min_block[iy] = d_left_weight[node_i];
        ////                        }
        //            d_excess_flow[node_i] = d_min_block[iy];
        ////            d_right_weight[node_i - 1] -= d_min_block[iy];
        ////            d_left_weight[node_i] += d_min_block[iy];
        //        }
        //        if(ix > 1 && ix < width && ix % threadPerBlock_x == 0)
        //        {
        //            d_excess_flow[node_i] = d_min_block[iy];
        ////            d_excess_flow[node_i + threadPerBlock_x/2] = d_min_block[iy];
        //        }

        // min1
        ix > 0 && ix < width-1 ? smem[idx] = d_up_right_sum[node_i] : smem[idx] = 1<<20;
        __syncthreads();
        for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
            if(threadIdx.x < stride){
                smem[idx] > smem[idx+stride] ? smem[idx] = smem[idx+stride] : 0;
            }
            __syncthreads();
        }
        min = smem[threadIdx.y * threadPerBlock_x];
        __syncthreads();
        if(d_up_right_sum[node_i] == min){
            //            d_excess_flow[iy*width + blockIdx.x*blockDim.x] = min/8;
            //            if(d_left_weight[node_i] !=0 || d_down_weight[node_i] != 0)
            //            {
            if(ix < width-1 /*&& d_left_weight[node_i] != 0*/)
            {
                //                d_excess_flow[node_i + 1] = d_right_weight[node_i];
                //                d_left_weight[node_i + 1] += d_right_weight[node_i];
                d_right_weight[node_i] = 0;
            }
            if(iy > 0 /*&& d_down_weight[node_i] != 0*/)
            {
                //                d_excess_flow[node_i - width] = d_up_weight[node_i];
                //                d_down_weight[node_i - width] = d_up_weight[node_i]*2;
                d_up_weight[node_i] = 0;
            }
            d_excess_flow[node_i] = min;
            //            }
        }
        __syncthreads();

        // min2
        ix > 0 && ix < width-1 ? smem[idx] = d_down_right_sum[node_i] : smem[idx] = 1<<20;
        __syncthreads();

        for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
            if(threadIdx.x < stride){
                smem[idx] > smem[idx+stride] ? smem[idx] = smem[idx+stride] : 0;
            }
            __syncthreads();
        }
        min = smem[threadIdx.y * threadPerBlock_x];
        __syncthreads();
        if(d_down_right_sum[node_i] == min){
            //            if(d_left_weight[node_i] !=0 || d_up_weight[node_i] != 0)
            //            {
            if(ix < width-1)
            {
                //                d_excess_flow[node_i + 1] = d_right_weight[node_i];
                //                d_left_weight[node_i + 1] = d_right_weight[node_i]*2;
                d_right_weight[node_i] = 0;
            }
            if(iy < height - 1)
            {
                //                d_excess_flow[node_i + width] = d_down_weight[node_i];
                //                d_up_weight[node_i + width] = d_down_weight[node_i]*2;
                d_down_weight[node_i] = 0;
            }
            //                d_right_weight[node_i] = 0;
            //                d_down_weight[node_i] = 0;
            //                d_left_weight[node_i] += d_left_weight[node_i];
            //                d_up_weight[node_i] += d_up_weight[node_i];
            //            }

            //            d_excess_flow[node_i] = min;
        }

        //                // min3
        //                ix > 0 && ix < width-1 ? smem[idx] = d_up_left_sum[node_i] : smem[idx] = 1<<20;
        //                __syncthreads();

        //                for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
        //                    if(threadIdx.x < stride){
        //                        smem[idx] > smem[idx+stride] ? smem[idx] = smem[idx+stride] : 0;
        //                    }
        //                    __syncthreads();
        //                }
        //                min = smem[threadIdx.y * threadPerBlock_x];
        //                __syncthreads();
        //                if(d_up_left_sum[node_i] == min){
        //                    //        d_up_weight[node_i] = 0;
        //                    //        d_left_weight[node_i] = 0;
        //                        if(iy > 0)
        //                            if(d_up_weight[node_i - width] !=0)
        //                                d_down_weight[node_i - width] = 0;
        //                        if(node_i > 0)
        //                            if(d_left_weight[node_i - 1] != 0)
        //                                d_right_weight[node_i - 1] = 0;

        //                    //                        d_excess_flow[node_i] = min;
        //                }


        //                // min4
        //                ix > 0 && ix < width-1 ? smem[idx] = d_down_left_sum[node_i] : smem[idx] = 1<<20;
        //                __syncthreads();

        //                for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
        //                    if(threadIdx.x < stride){
        //                        smem[idx] > smem[idx+stride] ? smem[idx] = smem[idx+stride] : 0;
        //                    }
        //                    __syncthreads();
        //                }
        //                min = smem[threadIdx.y * threadPerBlock_x];
        //                __syncthreads();
        //                if(d_down_left_sum[node_i] == min){
        //                    //        d_down_weight[node_i] = 0;
        //                    //        d_left_weight[node_i] = 0;
        //                        if(iy < height - 1)
        //                            if(d_down_weight[node_i + width] != 0)
        //                                d_up_weight[node_i + width] = 0;
        //                        if(node_i > 0)
        //                            if(d_left_weight[node_i - 1] != 0)
        //                                d_right_weight[node_i - 1] = 0;

        //                    //                        d_excess_flow[node_i] = min;
        //                }
    }
}
__global__ void
adjustGraph_kernel2(int *d_excess_flow, int *d_left_weight, int *d_right_weight, int *d_down_weight, int*d_up_weight,
                    int *d_graph_height, int *d_up_right_sum, int *d_up_left_sum, int *d_down_right_sum, int *d_down_left_sum,
                    int width, int height, int N){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int node_i = iy*width + ix;
    int idx = (threadIdx.y * threadPerBlock_x) + threadIdx.x;
    if(ix % threadPerBlock_x == 0 && ix != 0)
    {
        //        d_excess_flow[node_i] = d_right_weight[node_i - 1];
        d_right_weight[node_i - 1] = 0;
        d_left_weight[node_i] = 0;
    }
    if(iy % threadPerBlock_y == 0 && iy != 0)
    {
        d_up_weight[node_i] = 0;
        d_down_weight[node_i - width] = 0;
    }
}

__global__ void
adjustGraph_kernel3(int *d_excess_flow, int *d_left_weight, int *d_right_weight, int *d_down_weight, int*d_up_weight,
                    int *d_left_weight1, int *d_right_weight1, int *d_down_weight1, int*d_up_weight1,
                    int width, int height, int N){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int node_i = iy*width + ix;
    int idx = (threadIdx.y * threadPerBlock_x) + threadIdx.x;
    //    if(ix >= threadPerBlock_x && ix % threadPerBlock_x != 0) d_excess_flow[node_i] = 0;
    if(ix % threadPerBlock_x == 0 && ix != 0)
    {
        //        d_right_weight[node_i - 1] = 0;
        d_right_weight[node_i - 1] = d_right_weight1[node_i - 1];
        d_left_weight[node_i] = d_left_weight1[node_i]*2;
    }
    if(iy % threadPerBlock_y == 0 && iy != 0)
    {
        d_up_weight[node_i] = d_up_weight1[node_i];
        d_down_weight[node_i - width] = d_down_weight1[node_i - width];
    }
}

int CudaCut::cudaCutsSetupGraph(cv::Mat& img1, cv::Mat& img2){
    //    int x = 40;
    //    int y = (40-x)/2;
    cv::Mat area1 (img1, cv::Rect(img1.cols-OVERLAP_WIDTH, 0, OVERLAP_WIDTH, img1.rows));
    cv::Mat area2 (img2, cv::Rect(0, 0, OVERLAP_WIDTH, img2.rows));

    cv::Mat m1, m2;
    area1.convertTo(m1,CV_8UC1);
    area2.convertTo(m2,CV_8UC1);
    int xoffset = img1.cols - OVERLAP_WIDTH;
    //    if (img1.channels() == 3)
    //    {
    //        for(int y = 0; y < height; y++)
    //        {
    //            for(int x = 0; x < width - 1; x++)
    //            {
    //                Vec3b a0 = image1.at<Vec3b>(y, xoffset + x);
    //                Vec3b b0 = image2.at<Vec3b>(y, x);
    //                double cap0 = norm(a0, b0);

    //                Vec3b a1 = image1.at<Vec3b>(y, xoffset + x+1);
    //                Vec3b b1 = image2.at<Vec3b>(y, x + 1);
    //                double cap1 = norm(a1, b1);

    //                d_horizontal[y * (width+1) + x+1] = (int)(cap0 + cap1);
    //                //            horizontal[y * graph.width + x] = 0;
    //            }

    //            d_horizontal[y * (width+1) + width] = 0;
    //            d_horizontal[y * (width+1)] = 0;
    //        }

    //        for(int x = 0; x < width; x++)
    //        {
    //            for(int y = 0; y < height - 1; y++)
    //            {
    //                Vec3b a0 = image1.at<Vec3b>(y, xoffset + x);
    //                Vec3b b0 = image2.at<Vec3b>(y, x);
    //                double cap0 = norm(a0, b0);

    //                Vec3b a1 = image1.at<Vec3b>(y + 1, xoffset + x);
    //                Vec3b b1 = image2.at<Vec3b>(y + 1, x);
    //                double cap1 = norm(a1, b1);
    //                d_vertical[(y+1) * width + x] = (int)(cap0 + cap1);
    //            }
    //            d_vertical[(height) * width + x] = 0;
    //            d_vertical[x] = 0;
    //        }
    //    }
    if(img1.channels() == 1)
    {
        for(int y = 0; y < height; y++)
        {
            for(int x = 0; x < width - 1; x++)
            {
                uchar a0 = img1.at<uchar>(y, xoffset + x);
                uchar b0 = img2.at<uchar>(y, x);
                uchar cap0 = abs(a0 - b0);

                uchar a1 = img1.at<uchar>(y, xoffset + x+1);
                uchar b1 = img2.at<uchar>(y, x + 1);
                uchar cap1 = abs(a1 - b1);

                d_horizontal[y * (width+1) + x+1] = (int)(cap0 + cap1);
            }

            d_horizontal[y * (width+1) + width] = 0;
            d_horizontal[y * (width+1)] = 0;
        }

        for(int x = 0; x < width; x++)
        {
            for(int y = 0; y < height - 1; y++)
            {
                uchar a0 = img1.at<uchar>(y, xoffset + x);
                uchar b0 = img2.at<uchar>(y, x);
                uchar cap0 = abs(a0 - b0);

                uchar a1 = img1.at<uchar>(y + 1, xoffset + x);
                uchar b1 = img2.at<uchar>(y + 1, x);
                uchar cap1 = abs(a1 - b1);
                d_vertical[(y+1) * width + x] = (int)(cap0 + cap1);
                //            vertical[y * graph.width + x] = 0;
            }
            d_vertical[(height) * width + x] = 0;
            d_vertical[x] = 0;
        }
    }

    memcpy(h_m1, m1.ptr(0), sizeof(unsigned char)*graph_size);
    memcpy(h_m2, m2.ptr(0), sizeof(unsigned char)*graph_size);

    dim3 block(16, 8, 1);
    dim3 grid((width+block.x-1)/block.x, (height+block.y-1)/block.y, 1);
    dim3 block1(20, 8, 1);
    dim3 grid1((width+block.x-1)/block.x, (height+block.y-1)/block.y, 1);
    dim3 block2(1, 480, 1);
    dim3 grid2((width+block.x-1)/block.x, (height+block.y-1)/block.y, 1);
    setupGraph_kernel<<<grid, block>>>(d_horizontal, d_vertical, d_right_weight, d_left_weight, d_up_weight,
                                       d_down_weight, d_excess_flow, d_push_block_position, d_graph_height,
                                       d_relabel_mask, width, height, graph_size);
    adjustGraph_kernel<<<grid, block>>>(d_excess_flow, d_left_weight,d_right_weight, d_down_weight, d_up_weight, d_graph_height,
                                        d_up_right_sum, d_up_left_sum, d_down_right_sum, d_down_left_sum, width, height, graph_size);
//    gpuErrChk(hipMemcpy(h_right_weight, d_right_weight, sizeof(int)*graph_size, hipMemcpyDeviceToHost));
    //    gpuErrChk(hipMemcpy(h_left_weight, d_left_weight, sizeof(int)*graph_size, hipMemcpyDeviceToHost));
//    gpuErrChk(hipMemcpy(h_down_weight, d_down_weight, sizeof(int)*graph_size, hipMemcpyDeviceToHost));
    //    gpuErrChk(hipMemcpy(h_up_weight, d_up_weight, sizeof(int)*graph_size, hipMemcpyDeviceToHost));
    //    hipDeviceSynchronize();
    //        gpuErrChk(hipMemcpy(h_excess_flow, d_excess_flow, sizeof(int)*graph_size, hipMemcpyDeviceToHost));

    //        gpuErrChk(hipMemcpy(h_up_right_sum, d_up_right_sum, sizeof(int)*graph_size, hipMemcpyDeviceToHost));
    //        gpuErrChk(hipMemcpy(h_up_left_sum, d_up_left_sum, sizeof(int)*graph_size, hipMemcpyDeviceToHost));
//    writeToFile("h_right_weight_initial.txt", d_right_weight, width, height);
    //        writeToFile("../variable/h_left_weight_initial.txt", d_left_weight, width, height);
//    writeToFile("h_down_weight_initial.txt", d_down_weight, width, height);
    //        writeToFile("../variable/h_up_weight_initial.txt", d_up_weight, width, height);
    //        writeToFile("../variable/h_excess_flow_initial.txt", h_excess_flow, width, height);
    //        writeToFile("../variable/h_horizontal_initial.txt", d_horizontal, width+1, height);
    //        writeToFile("../variable/h_vertical_initial.txt", d_vertical, width, height+1);
    //        writeToFile("../variable/h_graph_height_initial.txt", d_graph_height, width, height);
    //        writeToFile("../variable/h_up_right_sum_initial.txt", h_up_right_sum, width, height);
    //        while(getchar() != 32);

    //    findMin(d_right_weight, d_min_block, width, height);
    //    writeToFile("d_min_block.txt", d_min_block,1, height);

    //    findMinCol(d_down_weight, d_min_col, width, height);
    //    writeToFile("d_min_col.txt", d_min_col,width, 1);

    //    adjustGraph_kernel1<<<grid, block>>>(d_excess_flow, d_left_weight,d_right_weight,
    //                                         d_down_weight, d_up_weight, d_graph_height,
    //                                         d_up_right_sum, d_up_left_sum, d_down_right_sum,
    //                                         d_down_left_sum, width, height, graph_size, d_min_block);
    //    hipDeviceSynchronize();
    //    writeToFile("d_min_block2.txt", d_min_block,1, height);

    //    adjustGraph_kernel2<<<grid, block>>>(d_excess_flow, d_left_weight,d_right_weight, d_down_weight,
    //                                         d_up_weight, d_graph_height, d_up_right_sum, d_up_left_sum,
    //                                         d_down_right_sum, d_down_left_sum, width, height, graph_size);

//    graphCorrectionImage(data_, d_left_weight,d_right_weight, d_down_weight, d_up_weight);
//    cv::Mat image(480, 80, CV_8UC1, data_);
//    cv::imshow("image", image);
//    cv::imwrite("image.png", image);
//    cv::waitKey();
    return 0;
}

int CudaCut::cudaCutsInit(){
    h_mem_init();
    d_mem_init();
    return 0;
}


void CudaCut::cudaCutsAtomic(int blockDimy, int number_loops, int backwardCycle,
                             int relabelCycle, int averageDistance, int stopPoint){
    //size = 80*480
    printf("%d-%d-%d-%d\n", backwardCycle, relabelCycle, averageDistance, stopPoint);
    dim3 block(16, blockDimy, 1);
    dim3 grid((width+block.x-1)/block.x, (height+block.y-1)/block.y, 1);
    int h_finished_count;

    int *d_finished_count;


    //hipMallocManaged((void**)&d_finished_count, sizeof(int));
    gpuErrChk(hipMalloc((void**)&d_finished_count, sizeof(int)));
    h_finished_count = 1;
    int counter = 0;
    int sum;
    while(h_finished_count != 0){



        push_kernel<<<grid, block>>>(d_right_weight, d_left_weight, d_up_weight, d_down_weight,
                                     d_excess_flow, d_graph_height, d_relabel_mask, d_height_backup,
                                     width, height, graph_size);
        if((counter+1) % backwardCycle == 0){ // c
            gpuErrChk(hipDeviceSynchronize());
            auto start1 = getMoment;
            memset(h_bfs_counter, 0, sizeof(int)*graph_size);
            for(int i = 0; i < graph_size; i++){
                if((i+1)%width != 0)
                    h_visited_backward[i] = false;
                else
                    h_visited_backward[i] = true;
            }
            globalRelabelCpu(d_right_weight, d_left_weight, d_down_weight, d_up_weight,
                             h_visited_backward, d_graph_height, h_bfs_counter, d_excess_flow);
            auto end1 = getMoment;
            std::cout << "Global Relabel Time = "<< TimeCpu(end1, start1)/1000.0 << "\n";
        }

        if(counter <= 2000)
        {
            h_finished_count = 0;
            gpuErrChk(hipMemcpy(d_finished_count, &h_finished_count, sizeof(int), hipMemcpyHostToDevice));
            check_finished_condition<<<grid, block>>>(d_excess_flow, d_finished_count, d_graph_height, width, height, graph_size);
            gpuErrChk(hipMemcpy(&h_finished_count, d_finished_count, sizeof(int), hipMemcpyDeviceToHost));
            h_active_node[counter] = counter;
            h_active_node[counter + 2000] =  h_finished_count;
            if(counter >= averageDistance)
            {
                int sum = 0;
                for(int i = 0; i < averageDistance; i++)
                {
                    sum += h_active_node[counter - i + 2000];
                }
                sum = sum/averageDistance;
                h_average_active[counter - averageDistance] = counter - averageDistance;
                h_average_active[counter - averageDistance + 2000] = sum;
                if(sum <= stopPoint) break;
            }
//            h_finished_count = 1;
        }

        if((counter)%relabelCycle == 0){
            if((counter) % 2 == 0)
            {
                h_finished_count = 0;
                gpuErrChk(hipMemcpy(d_finished_count, &h_finished_count, sizeof(int), hipMemcpyHostToDevice));
                check_finished_condition<<<grid, block>>>(d_excess_flow, d_finished_count, d_graph_height, width, height, graph_size);
                gpuErrChk(hipMemcpy(&h_finished_count, d_finished_count, sizeof(int), hipMemcpyDeviceToHost));
//                gpuErrChk(hipDeviceSynchronize());
//                std::cout << "number active nodes " << h_finished_count << std::endl;
            }

            relabel_kernel<<<grid, block>>>(d_right_weight, d_left_weight, d_up_weight, d_down_weight,
                                            d_graph_height, d_relabel_mask, d_height_backup,
                                            d_excess_flow, width, height, graph_size);
            //gpuErrChk(hipDeviceSynchronize());
            //}
            //                std::cout << "number active nodes " << h_finished_count << std::endl;
            //h_active_node[counter/2] = counter;
            //h_active_node[counter/2 + 3000] =  *h_finished_count;
            //            graphCorrectionImage(data_, d_left_weight,d_right_weight, d_down_weight, d_up_weight);
            //            cv::Mat image(480, 80, CV_8UC1, data_);
            //            cv::imshow("image", image);
            //            cv::imwrite("image.png", image);
            //            cv::waitKey();
            //            if(counter <= 1200)
            //            {
            ////                std::cout << "number active nodes " << h_finished_count << std::endl;
            //                                h_active_node[counter/2] = counter;
            //                                h_active_node[counter/2 + 600] =  h_finished_count;
            //            }
        }

        counter++;
        if(counter == number_loops)
            h_finished_count = 0;
    }
    gpuErrChk(hipDeviceSynchronize());
    //    graphCorrectionImage(data_, d_left_weight,d_right_weight, d_down_weight, d_up_weight);
    //    cv::Mat image(480, 80, CV_8UC1, data_);
    //    cv::imshow("image1", image);
    //    cv::imwrite("image1.png", image);
    //    cv::waitKey();
    //    //vec1.push_back(t);
    //    //std::cout << "kernel Time = "<< std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000 << std::endl;
        hipFree(d_finished_count);
    cout << "couter = " << counter << endl;
    writeToFile("h_active_node.txt", h_active_node, 2000, 2);
//        writeToFile("h_average_active.txt", h_average_active, 2000, 2);

    //    gpuErrChk(hipMemcpy(h_excess_flow, d_excess_flow, sizeof(int)*graph_size, hipMemcpyDeviceToHost));
    //    //gpuErrChk(hipDeviceSynchronize());
    //    sum = 0;
    //    for(int i = 0; i < graph_size; i++){
    //        if((i+1)%width == 0)
    //            sum += h_excess_flow[i];
    //    }


    //    std::cout << "max flow: " << sum << std::endl;
    //    std::cout << "final_excess_flow " << std::endl;
    //    std::cout << "\n";


}

int CudaCut::cudaCutsAtomicOptimize(cv::Mat& result)
{
    //cudaCutsAtomic(result);
    //bfsLabeling();

    return 0 ;

}

void CudaCut::cudaCutsFreeMem()
{

    free(h_left_weight);
    free(h_right_weight);
    free(h_down_weight);
    free(h_up_weight);

    free(h_excess_flow);
    free(h_relabel_mask);
    free(h_graph_height);
    free(h_height_backup);
    free(h_visited);
    free(h_frontier);
    free(h_visited_backward);
    free(h_visited_forward);

    free(h_m1);
    free(h_m2);
    free(h_process_area);
    free(h_horizontal);
    free(h_vertical);
    free(h_bfs_counter);
    free(h_active_node);
    free(data_);
    free(h_average_active);




    gpuErrChk(hipFree(d_left_weight));
    gpuErrChk(hipFree(d_right_weight));
    gpuErrChk(hipFree(d_down_weight));
    gpuErrChk(hipFree(d_up_weight));
    gpuErrChk(hipFree(d_min_block));
    gpuErrChk(hipFree(d_min_col));
    gpuErrChk(hipFree(d_excess_flow));
    gpuErrChk(hipFree(d_relabel_mask));
    gpuErrChk(hipFree(d_graph_height));
    gpuErrChk(hipFree(d_height_backup));
    //    gpuErrChk(hipFree(d_visited));
    //    gpuErrChk(hipFree(d_frontier));

    //    gpuErrChk(hipFree(d_m1));
    //    gpuErrChk(hipFree(d_m2));
    //    gpuErrChk(hipFree(d_process_area));
    //    gpuErrChk(hipFree(d_horizontal));
    //    gpuErrChk(hipFree(d_vertical));
    //    gpuErrChk(hipFree(d_push_block_position));
    gpuErrChk(hipFree(d_up_right_sum));
    gpuErrChk(hipFree(d_up_left_sum));
    gpuErrChk(hipFree(d_down_right_sum));
    gpuErrChk(hipFree(d_down_left_sum));
    gpuErrChk(hipFree(d_bfs_counter));

}

